//  nvcc -arch=sm_50 strassen-cuda_v1.cu -o strassen-cuda_v1 -lcublas 
//  bash ./mybash
//
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16
#define N_TEST 50
#define MAX_DEPTH 20


float *temp_A, *temp_B, *mul_Array;
float *mat_A[MAX_DEPTH], *mat_B[MAX_DEPTH], *mat_C[MAX_DEPTH];
float *M1[MAX_DEPTH], *M2[MAX_DEPTH], *M3[MAX_DEPTH], *M4[MAX_DEPTH], *d_M5[MAX_DEPTH], *M6[MAX_DEPTH], *M7[MAX_DEPTH];


template <typename matrixArray>
void fillMatrix(matrixArray* arr, const int N)
{
	for (int i = 0; i < N; ++i)
	{
		arr[i] = (matrixArray) (rand() & 0xF);
	}
}

template <typename matrixArray>
__global__
void classicalMatmul(matrixArray* A, matrixArray* B, matrixArray* C, const int dim)
{
	const int row = blockIdx.y * blockDim.y + threadIdx.y;
	const int col = blockIdx.x * blockDim.x + threadIdx.x;
	const int gd = gridDim.x;

	__shared__ matrixArray _A[BLOCK_SIZE][BLOCK_SIZE], _B[BLOCK_SIZE][BLOCK_SIZE];

	if (row < dim && col < dim)
	{
		matrixArray sum = 0;
		for (int k = 0; k < gd; ++k)
		{
			_A[threadIdx.y][threadIdx.x] = A[row*dim + k*BLOCK_SIZE + threadIdx.x];
			_B[threadIdx.y][threadIdx.x] = B[(k*BLOCK_SIZE+threadIdx.y) * dim + col];
			__syncthreads();

			for (int l = 0; l < BLOCK_SIZE; ++l)
			{
				sum += _A[threadIdx.y][l] * _B[l][threadIdx.x];
			}
			__syncthreads();
		}

		C[row*dim + col] = sum;
	}
}


template <typename matrixArray>
void strassenMatmul(hipblasHandle_t& handle, matrixArray* A, matrixArray* B, matrixArray* C, const int dim, const int d, const int threshold)
{
	const int dim_2 = dim/2;

	int lda = dim, ldb = dim, ldc = dim_2;
	int m = dim_2, n = dim_2;
	matrixArray one = 1, zero = 0, m_one = -1;

	if (dim <= threshold)
	{
		dim3 block(BLOCK_SIZE, BLOCK_SIZE);
		dim3 grid((dim+BLOCK_SIZE-1)/BLOCK_SIZE, (dim+BLOCK_SIZE-1)/BLOCK_SIZE);
		classicalMatmul<matrixArray><<< grid, block >>>(A, B, C, dim);
		// cublasSgemm(handle, CUBLAS_OP_N, CUBLAS_OP_N, dim, dim, dim, &one, B, dim, A, dim, &zero, C, dim);
		return;
	}


	/* M1 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2*dim+dim_2, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2*dim+dim_2, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M1[d+1], dim_2, d+1, threshold);

	/* M2 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &one, A+dim_2*dim+dim_2, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &zero, B, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M2[d+1], dim_2, d+1, threshold);

	/* M3 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &zero, A, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2, lda, &m_one, B+dim_2*dim+dim_2, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M3[d+1], dim_2, d+1, threshold);

	/* M4 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim+dim_2, lda, &zero, A, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &m_one, B, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M4[d+1], dim_2, d+1, threshold);

	/* M5 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A, lda, &one, A+dim_2, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim+dim_2, lda, &zero, B, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], d_M5[d+1], dim_2, d+1, threshold);

	/* M6 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2*dim, lda, &m_one, A, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B, lda, &one, B+dim_2, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M6[d+1], dim_2, d+1, threshold);

	/* M7 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, A+dim_2, lda, &m_one, A+dim_2*dim+dim_2, ldb, mat_A[d+1], ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, B+dim_2*dim, lda, &one, B+dim_2*dim+dim_2, ldb, mat_B[d+1], ldc);
	strassenMatmul(handle, mat_A[d+1], mat_B[d+1], M7[d+1], dim_2, d+1, threshold);


	/* C1 */
	lda = dim, ldb = dim/2, ldc = dim;  // C = C + B
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C, lda, &one, M1[d+1], ldb, C, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, M4[d+1], ldb, C, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &m_one, d_M5[d+1], ldb, C, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C, lda, &one, M7[d+1], ldb, C, ldc);

	/* C2 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2, lda, &one, M3[d+1], ldb, C+dim_2, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2, lda, &one, d_M5[d+1], ldb, C+dim_2, ldc);

	/* C3 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim, lda, &one, M2[d+1], ldb, C+dim_2*dim, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim, lda, &one, M4[d+1], ldb, C+dim_2*dim, ldc);

	/* C4 */
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &zero, C+dim_2*dim+dim_2, lda, &one, M1[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &m_one, M2[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, M3[d+1], ldb, C+dim_2*dim+dim_2, ldc);
	hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, &one, C+dim_2*dim+dim_2, lda, &one, M6[d+1], ldb, C+dim_2*dim+dim_2, ldc);
}

int main(int argc, char** argv)
{
	if (argc != 3)
	{
		printf("Usage: %s <size> <threshold>\n", argv[0]);
		exit(0);
	}

    clock_t t1, t2;
	/* Initialize */

	int n_size = atoi(argv[1]);
	int threshold = atoi(argv[2]);
	//int check = atoi(argv[3]);
	
	printf("[size= %d, threshold = %d] \n", n_size, threshold );

	dim3 block(BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((n_size+BLOCK_SIZE-1)/BLOCK_SIZE, (n_size+BLOCK_SIZE-1)/BLOCK_SIZE);

	assert(n_size >= threshold && threshold >= BLOCK_SIZE);

	size_t nBytes = n_size * n_size * sizeof(float);

	temp_A = (float*) malloc(nBytes);
	temp_B = (float*) malloc(nBytes);
	mul_Array = (float*) malloc(nBytes);

	srand(0);
	fillMatrix<float>(temp_A, n_size*n_size);
	fillMatrix<float>(temp_B, n_size*n_size);

	int depth, _dim = n_size;
	for (depth = 0; depth < MAX_DEPTH && _dim > 0; ++depth)
	{
		hipMalloc((float**) &mat_A[depth], _dim*_dim*sizeof(float));
		hipMalloc((float**) &mat_B[depth], _dim*_dim*sizeof(float));

		if (depth == 0)
		{
			hipMalloc((float**) &mat_C[depth], _dim*_dim*sizeof(float));
		}
		else
		{
			hipMalloc((float**) &M1[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &M2[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &M3[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &M4[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &d_M5[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &M6[depth], _dim*_dim*sizeof(float));
			hipMalloc((float**) &M7[depth], _dim*_dim*sizeof(float));
		}
		_dim /= 2;
	}

	hipMemcpy(mat_A[0], temp_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(mat_B[0], temp_B, nBytes, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);


	/* Run classicalMatmul */
	t1 = clock();
	for (int i = 0; i < N_TEST; ++i)
	{
		classicalMatmul<float><<< grid, block >>>(mat_A[0], mat_B[0], mat_C[0], n_size);
		hipDeviceSynchronize();
	}
	t2 = clock();
	printf("[classicalMatmul] %lfs \n", (t2-t1)/(double)(CLOCKS_PER_SEC));


	/* Run strassenMatmul */
	t1 = clock();
	for (int i = 0; i < N_TEST; ++i)
	{
		strassenMatmul<float>(handle, mat_A[0], mat_B[0], mat_C[0], n_size, 0, threshold);
	}
	t2 = clock();
	printf("[strassenMatmul] %lfs \n\n", (t2-t1)/(double)(CLOCKS_PER_SEC));
	hipMemcpy(mul_Array, mat_C[0], nBytes, hipMemcpyDeviceToHost);


	/* Free memory */
	hipblasDestroy(handle);

	for (int i = 0; i < depth; ++i)
	{
		hipFree(mat_A[i]);
		hipFree(mat_B[i]);

		if (i == 0)
		{
			hipFree(mat_C[i]);
		}
		else
		{
			hipFree(M1[i]);
			hipFree(M2[i]);
			hipFree(M3[i]);
			hipFree(M4[i]);
			hipFree(d_M5[i]);
			hipFree(M6[i]);
			hipFree(M7[i]);
		}
	}

	hipDeviceReset();

	free(temp_A);
	free(temp_B);
	free(mul_Array);

	return 0;
}